#include <time.h>
#include <stdio.h>

#include "ttime.h"

static hipEvent_t start, stop;

static struct timespec cpu_start, cpu_stop;

bool inited = false;

void start_time_cuda() {
    if (!inited) {
        inited = true;
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }
    hipEventRecord( start, 0 );
}


void start_time_cpu() {
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);
}

void stop_time_cuda() {
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, stop);
  printf("Total GPU execution time:  %3.1f ms\n", elapsedTime);
}


void stop_time_cpu() {
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
  double result = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
  printf( "CPU execution time:  %3.1f ms\n", result);
}
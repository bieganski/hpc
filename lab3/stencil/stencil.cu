
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#define RADIUS        30
#define NUM_ELEMENTS  1000000

#define NUM_THREADS_PER_BLOCK  32 

static void handleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define cudaCheck( err ) (handleError(err, __FILE__, __LINE__ ))


__device__ __host__ int in_range(int idx) {
  if (idx >= 0 && idx <= NUM_ELEMENTS -1)
    return 1;
  else
    return 0;
}

__device__ __host__ void algo(int i, int *in, int *out) {
  for(int j = 0; j <= RADIUS; j++) {
    int idx1 = i - j;
    if (in_range(idx1))
      out[i] += in[idx1];
    if (j == 0)
      continue;
    int idx2 = i + j;
    if (!in_range(idx2))
      out[i] += in[idx2];
  }
}


__global__ void stencil_1d(int *in, int *out) {
  int i = threadIdx.x + (blockIdx.x * blockDim.x);
  // int all = blockDim.x * gridDim.x;
  if (i == 0)
    printf("ZERO\n");
  if(i == NUM_ELEMENTS - 1)
    printf("MAX\n");
  algo(i, in, out);
}

void cpu_stencil_1d(int *in, int *out) {
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    algo(i, in, out);
  }
}

int compare(int * r1, int * r2) {
  int diffs = 0;
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    if (r1[i] != r2[i])
    diffs++;
  }
  return diffs;
}

int main() {
  //PUT YOUR CODE HERE - INPUT AND OUTPUT ARRAYS
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start, 0 );

  int *dev_in, *dev_out;

  int BYTES_NUM = sizeof(int) * NUM_ELEMENTS;

  int *host_in = (int*)malloc(BYTES_NUM);
  int *host_out = (int*)malloc(BYTES_NUM);

  int *from_dev_out = (int*)malloc(BYTES_NUM);

  //PUT YOUR CODE HERE - DEVICE MEMORY ALLOCATION
  hipMalloc((void**)&dev_in, BYTES_NUM);
  hipMalloc((void**)&dev_out, BYTES_NUM);

  //PUT YOUR CODE HERE - KERNEL EXECUTION
  for(int i = 0; i < NUM_ELEMENTS; i++) {
    host_in[i] = i;
  }

  hipMemcpy(dev_in, host_in, BYTES_NUM, hipMemcpyHostToDevice);
  
  int num_blocks = NUM_ELEMENTS / NUM_THREADS_PER_BLOCK + 1;

  stencil_1d<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(dev_in, dev_out);

  // blockDim.x,y,z gives the number of threads in a block, in the particular direction
  // gridDim.x,y,z gives the number of blocks in a grid, in the particular direction

  
  cudaCheck(hipPeekAtLastError());


  //PUT YOUR CODE HERE - COPY RESULT FROM DEVICE TO HOST
  hipMemcpy(from_dev_out, dev_out, BYTES_NUM, hipMemcpyDeviceToHost);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, stop);
  printf("Total GPU execution time:  %3.1f ms\n", elapsedTime);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //PUT YOUR CODE HERE - FREE DEVICE MEMORY  
  hipFree(dev_in);
  hipFree(dev_out);

  struct timespec cpu_start, cpu_stop;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);

  cpu_stencil_1d(host_in, host_out);

  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
  double result = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
  printf( "CPU execution time:  %3.1f ms\n", result);

  int diffs = compare(from_dev_out, host_out);
  if (diffs != 0)
    printf("BAD: diff: %d/%d\n", diffs, NUM_ELEMENTS);
  else
    puts("OK, 0 diffs!\n");

  
  return 0;
}

#include "hip/hip_runtime.h"
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <cassert>

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <set>
#include <float.h>

#include <thrust/device_vector.h>
#include <thrust/partition.h>
#include <thrust/sequence.h>
#include <thrust/distance.h>
#include <thrust/functional.h>

#include "hasharray.h"
#include "utils.h"

#include "modopt.h"


__host__
void contract(const uint32_t V_MAX_IDX,
                          uint32_t* __restrict__ V, 
                          uint32_t* __restrict__ E,
                          float*    __restrict__ W,
                          float*    __restrict__ k,
                          const uint32_t* __restrict__ comm,
                          thrust::device_vector<uint32_t>& globalCommAssignment);


using HA = HashArray;

// [ BINS[i], BINS[i+1] ) (right side exclusive)
__device__ uint32_t BINS[] =  
    {
        0, // [0,1) is handled separately (lonely nodes; their modularity impact is 0)
        4,
        8,
        16,
        32,
        96,
        320,
        1024,
        UINT32_MAX // hash arrays in global memory
    };

__global__
void computeAC(const uint32_t V_MAX_IDX,
                const float* __restrict__ k,
               float*       __restrict__ ac,
               const uint32_t*    __restrict__ comm) {

    int tid = 1 + threadIdx.x + (blockIdx.x * blockDim.x);
    if (tid > V_MAX_IDX)
        return;
    atomicAdd(&ac[comm[tid]], k[tid]);
}

__host__
void zeroAC(float* ac, uint32_t V_MAX_IDX) {
    std::memset(ac, 0, sizeof(float) * (V_MAX_IDX + 1));
}

__device__ 
__forceinline__ 
unsigned int __lane_id() { 
    unsigned int laneid; 
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(laneid)); 
    return laneid; 
}


// https://fgiesen.wordpress.com/2013/01/21/order-preserving-bijections/
__device__
__forceinline__
uint32_t int_to_uint(int32_t val) {
    return val ^ 0x80000000;
}

__device__
__forceinline__
int32_t uint_to_int(uint32_t val) {
    return val ^ 0x80000000;
}

__device__
__forceinline__
int32_t float_to_int(float f32_val) {
    int32_t tmp = __float_as_int(f32_val);
    return tmp ^ ((tmp >> 31) & 0x7fffffff);
}

__device__
__forceinline__
float int_to_float(int32_t i32_val) {
    int32_t tmp = i32_val ^ ((i32_val >> 31) & 0x7fffffff);
    return __int_as_float(tmp);
}

__device__
__forceinline__
float uint_to_float(uint32_t ui32_val) {
    int32_t tmp1 = ui32_val ^ 0x80000000;
    int32_t tmp2 = tmp1 ^ (tmp1 >> 31);
    return __uint_as_float(tmp2);
}

__device__
__forceinline__
uint32_t float_to_uint(float f32_val) {
    uint32_t tmp = __float_as_uint(f32_val);
    return tmp ^ (((int32_t)tmp >> 31) | 0x80000000);
}

#define VAR_MEM_PER_VERTEX_BYTES_DEFINE 16

/**
 * Version of 'reassign_nodes' that handles nodes with degree > 32.
 * It deduces whether total size of hasharrays fits into shared memory,
 * if no, then it uses global memory.
 */
__global__ 
void reassign_huge_nodes(
                        const uint32_t  numNodes,
                        const uint32_t* __restrict__ binNodes,
                        const uint32_t* __restrict__ V,
                        const uint32_t* __restrict__ E,
                        const float*    __restrict__ W,
                        const float*    __restrict__ k,
                        const float*    __restrict__ ac,
                        const uint32_t* __restrict__ comm,
                        uint32_t*       __restrict__ newComm,
                        const uint32_t maxDegree,
                        const uint32_t nodesPerBlock,
                        const uint32_t hasharrayEntries,
                        const float m,
                        const void*    globalHasharray) {

    extern __shared__ char shared_mem[]; // shared memory is one-byte char type, for easy offset applying

    int i_ptr = threadIdx.y + blockIdx.x * blockDim.y; // my vertex pointer
    int edge_ptr = threadIdx.x; // my edge pointer

    assert(edge_ptr < maxDegree);

    KeyValueFloat* hashWeight;
    KeyValueInt*   hashComm;

    // TODO: customize these
    // VARIABLES:  ei_to_Ci, loop, deltaModRes
    uint32_t VAR_MEM_PER_VERTEX_BYTES = sizeof(float) + sizeof(float) + sizeof(uint64_t);

    assert(VAR_MEM_PER_VERTEX_BYTES_DEFINE == VAR_MEM_PER_VERTEX_BYTES);

    uint32_t COMMON_VARS_SIZE_BYTES = VAR_MEM_PER_VERTEX_BYTES * nodesPerBlock;

    // TODO customize this
    bool shared = globalHasharray == nullptr;

    // very careful pointer handling here, because of lots of type mismatches and casting
    if (shared) {
        uint32_t off = COMMON_VARS_SIZE_BYTES + (i_ptr % nodesPerBlock) * (2 * hasharrayEntries) * sizeof(KeyValueFloat);
        hashWeight = (KeyValueFloat*) (&shared_mem[off]);
        assert( shared_mem + COMMON_VARS_SIZE_BYTES <= (char*) hashWeight);
        hashComm   = (KeyValueInt*)   hashWeight + hasharrayEntries;
        assert(globalHasharray == nullptr);
    } else {
        hashWeight = ( (KeyValueFloat*) globalHasharray ) + i_ptr * (2 * hasharrayEntries);
        hashComm   = (KeyValueInt*)   hashWeight + hasharrayEntries;
    }

    // before any early return, let's utilize all threads for zeroing memory.

    uint64_t* __tmp = (uint64_t*) shared_mem;
    for (int i =0; i < COMMON_VARS_SIZE_BYTES / 8; i++) {
        __tmp[i] = 0;
    }

    for (int i = edge_ptr; i < hasharrayEntries; i += maxDegree) {
        hashWeight[i] = {.key = hashArrayNull, .value = (float) 0}; // 0 for easy atomicAdd
        hashComm[i]   = {.key = hashArrayNull, .value = hashArrayNull};
    }

    if (numNodes -1 < i_ptr) {
        return;
    }
    int i = binNodes[i_ptr]; // my vertex

    if (V[i + 1] - V[i] -1 < edge_ptr) {
        return;
    }
    uint32_t j = E[V[i] + edge_ptr]; // my neighbor

    // variables common for each vertex, accumulating ei_to_Ci value 
    // computed in parallel
    uint32_t ei_to_ci_off_bytes = (i_ptr % nodesPerBlock) * VAR_MEM_PER_VERTEX_BYTES;
    int32_t* glob_ei_to_Ci = (int32_t*) &shared_mem[ei_to_ci_off_bytes];
    
    uint32_t loop_off_bytes = ei_to_ci_off_bytes + sizeof(int32_t);
    assert(ei_to_ci_off_bytes < loop_off_bytes);
    assert(loop_off_bytes < COMMON_VARS_SIZE_BYTES);
    int32_t* glob_loop = (int32_t*) &shared_mem[loop_off_bytes];

    __syncthreads();

    // ok, data initialized, let the run start
    uint32_t mySlot = HA::insertInt(hashComm, comm[j], comm[j], hasharrayEntries);
    float sum = HA::addFloatSpecificPos(hashWeight, mySlot, W[V[i] + edge_ptr]);

    float loop = i == j ? W[V[i] + edge_ptr] : 0;
    float ei_to_Ci = comm[j] == comm[i] ? hashWeight[mySlot].value : 0;

    atomicMax(glob_loop, float_to_int(loop));
    atomicMax(glob_ei_to_Ci, float_to_int(ei_to_Ci));

    __syncthreads();

    if (edge_ptr == 0) {
        loop = int_to_float(*glob_loop);
        ei_to_Ci = int_to_float(*glob_ei_to_Ci);
        ei_to_Ci -= loop;
    }

    float deltaModRaw = comm[j] >= comm[i] ? 
        -(1 << 5) : 
        k[i] * ( ac[comm[i]] - k[i] - ac[comm[j]] ) / (2 * m * m)  +  hashWeight[mySlot].value / m;

    uint32_t newCommIdx = comm[j];

    // Now, we must perform reduction on deltaMod values, to find best newCommIdx.
    // It's kinda hacky, cause in case of obtaining two identical deltaMod values,
    // we choose community with lower idx. It is accomplished by finding maximal value
    // of pair (int(deltaMod), -newCommIdx). We implement it using concatenation
    // of two unsigned values (obtained by order-preserving bijection from floats).
    uint32_t deltaMod_off_bytes = loop_off_bytes + sizeof(float);
    uint64_t* glob_deltaMod = (uint64_t*) &shared_mem[deltaMod_off_bytes];
    assert(newCommIdx != UINT32_MAX);
    assert(newCommIdx != 0);
    uint32_t newCommIdxRepr = -1 - newCommIdx; // bits flipped

    assert(-1 - newCommIdxRepr == newCommIdx);

    uint64_t deltaMod = (((uint64_t) float_to_uint(deltaModRaw)) << 31) | newCommIdxRepr;

    assert((uint32_t)deltaMod != 0);

    assert(sizeof(uint64_t) == sizeof(unsigned long long int));

    atomicMax((unsigned long long int*) glob_deltaMod, (unsigned long long int)deltaMod);

    __syncthreads();

    if (edge_ptr == 0) {
        // TODO, commented one and line below aren't equivalent, it breaks for negative floats.
        // float deltaModBest = uint_to_float((uint32_t)(*glob_deltaMod >> 31));
        float deltaModBest = int_to_float(uint_to_int((uint32_t)(*glob_deltaMod >> 31)));
        uint64_t test = *glob_deltaMod;
        uint32_t commIdxBest = (uint32_t) -1 - (uint32_t) (test & UINT32_MAX);

        float gain = deltaModBest - ei_to_Ci / m;

        if (gain > 0 && commIdxBest < comm[i]) {
            assert(commIdxBest > 0);
            newComm[i] = commIdxBest;
        } else {
            newComm[i] = comm[i];
        }
    }
}

__global__ 
void reassign_nodes(
                        const uint32_t  numNodes,
                        const uint32_t* __restrict__ binNodes,
                        const uint32_t* __restrict__ V,
                        const uint32_t* __restrict__ E,
                        const float*    __restrict__ W,
                        const float*    __restrict__ k,
                        const float*    __restrict__ ac,
                        const uint32_t* __restrict__ comm,
                        uint32_t*       __restrict__ newComm,
                        const uint32_t maxDegree,
                        const uint32_t nodesPerBlock,
                        const uint32_t hasharrayEntries,
                        const float m) {

    extern __shared__ KeyValueFloat hashtables[];

    assert(next_2_pow(maxDegree) == maxDegree); // maxDegree is power of 2
    
    int i_ptr =  threadIdx.y + blockIdx.x * blockDim.y; // my node pointer
    int edgeNum = threadIdx.x; // my edge pointer

    if (numNodes -1 < i_ptr) {
        return;
    }

    int i = binNodes[i_ptr];

    if (V[i + 1] - V[i] -1 < edgeNum) {
        return;
    }

    uint32_t j = E[V[i] + edgeNum];

    // each hashtable contains of `hasharrayEntries` elements
    KeyValueFloat* hashWeight = (KeyValueFloat*) hashtables + (i_ptr % nodesPerBlock) * (2 * hasharrayEntries);
    KeyValueInt*   hashComm   = (KeyValueInt*)   hashWeight + hasharrayEntries;

    // TODO tu jest za dużo roboty
    for (int i = 0; i < hasharrayEntries; i++) {
        hashWeight[i].key = hashArrayNull;
        hashWeight[i].value = (float) 0;
        hashComm[i].key = hashArrayNull;
        hashComm[i].value = hashArrayNull;
    }
    __syncwarp();

    // ok, data initialized, let the run start
    uint32_t mySlot = HA::insertInt(hashComm, comm[j], comm[j], hasharrayEntries);
    float sum = HA::addFloatSpecificPos(hashWeight, mySlot, W[V[i] + edgeNum]);

    __syncwarp();

    uint32_t mask = __ballot_sync(FULL_MASK, edgeNum < maxDegree / 2);
    
    // sum of weights from node i to Ci\{i} from paper doesn't use loop values
    float loop = i == j ? W[V[i] + edgeNum] : 0;
    float ei_to_Ci = comm[j] == comm[i] ? hashWeight[mySlot].value : 0;

    float todo = i == j ? W[V[i] + edgeNum] : 0;

    __syncwarp();

    for (int offset = maxDegree / 2; offset > 0; offset /= 2) {
        ei_to_Ci = fmaxf(ei_to_Ci, __shfl_down_sync(mask, ei_to_Ci, offset)); // only warp with idx % maxDegree == 0 keeps proper value
        loop = fmaxf(loop, __shfl_down_sync(mask, loop, offset));

        // todo += __shfl_down_sync(mask, todo, offset);
    }
   

    // TODO important sanity check (asserts no multi-loops)
    // if (edgeNum == 0) {
    //     assert(todo == loop);
    //     ei_to_Ci -= loop;
    // }

    // lack of -(e_i -> C_i\{i} / m) addend in that sum, it will be computed later
    float deltaMod = comm[j] >= comm[i] ? 
        -(1 << 5) : 
        k[i] * ( ac[comm[i]] - k[i] - ac[comm[j]] ) / (2 * m * m)  +  hashWeight[mySlot].value / m;

    uint32_t newCommIdx = comm[j];

    for (int offset = maxDegree / 2; offset > 0; offset /= 2) {
        float deltaModRed = __shfl_down_sync(mask, deltaMod, offset);
        uint32_t newCommIdxRed = __shfl_down_sync(mask, newCommIdx, offset);

        if (newCommIdxRed == 0)
            continue; // TODO - brzydki hack na undefined behavior __shfl_down_sync

        if (deltaModRed > deltaMod) {
            deltaMod = deltaModRed;
            newCommIdx = newCommIdxRed;
        } else if (deltaModRed == deltaMod) {
            newCommIdx = (uint32_t) fminf((uint32_t) newCommIdx, (uint32_t) newCommIdxRed);
        }
    }

    if (edgeNum == 0) {
        float gain = deltaMod - ei_to_Ci / m;

        if (gain > 0 && newCommIdx < comm[i]) {
            assert(newCommIdx != 0);
            newComm[i] = newCommIdx;
        } else {
            newComm[i] = comm[i];
        }
    }
}

__host__ 
float reassign_communities_bin(
                        const uint32_t* binNodes,
                        const uint32_t binNodesNum,
                        const uint32_t* __restrict__ V, 
                        const uint32_t* __restrict__ E,
                        const float*    __restrict__ W,
                        const float*    __restrict__ k,
                        const float*    __restrict__ ac,
                        uint32_t* __restrict__ comm,
                        uint32_t* __restrict__ newComm,
                        const uint32_t maxDegree,
                        const float m) {

    // TODO customize this, maybe check 2 * maxDegree?
    uint32_t hashArrayEntriesPerComm;

    assert(sizeof(KeyValueFloat) == sizeof(KeyValueInt));

    uint32_t threadsX = maxDegree;
    uint32_t maxThreadsY = 1024 / threadsX;
    uint32_t threadsY = min(maxThreadsY, binNodesNum);
    uint32_t blockNum = ceil( (float)binNodesNum / threadsY );

    // assert(blockNum * threadsY >= binNodesNum); // TODO

    dim3 dim(maxDegree, threadsY);
    
    if (maxDegree == 1024 || maxDegree == UINT32_MAX) {
        hashArrayEntriesPerComm = next_2_pow(4096);
        
        float* globalHashArrays;
        HANDLE_ERROR(hipHostAlloc((void**)&globalHashArrays, sizeof(KeyValueFloat) * binNodesNum * (2 * hashArrayEntriesPerComm), hipHostMallocDefault));
        std::memset(globalHashArrays, '\0', sizeof(KeyValueFloat) * binNodesNum * (2 * hashArrayEntriesPerComm));
        HANDLE_ERROR(hipHostGetDevicePointer(&globalHashArrays, globalHashArrays, 0));

        uint32_t shmBytes = threadsY * VAR_MEM_PER_VERTEX_BYTES_DEFINE;
        assert(shmBytes <= SHARED_MEM_SIZE);

        reassign_huge_nodes<<<blockNum, dim, shmBytes>>> (binNodesNum, binNodes, 
            V, E, W, k, ac, comm, newComm, maxDegree, threadsY, hashArrayEntriesPerComm, m, globalHashArrays);

        hipDeviceSynchronize();
        HANDLE_ERROR(hipHostFree(globalHashArrays));
    } else {
        hashArrayEntriesPerComm = next_2_pow(maxDegree); // TODO koniecznie sprawdzic +1

        if (maxDegree <= WARP_SIZE) {
            uint32_t shmBytes = (2 * hashArrayEntriesPerComm) * sizeof(KeyValueInt) * threadsY;
            assert(shmBytes <= SHARED_MEM_SIZE);

            reassign_nodes<<<blockNum, dim, shmBytes>>>      (binNodesNum, binNodes, 
                V, E, W, k, ac, comm, newComm, maxDegree, threadsY, hashArrayEntriesPerComm, m);
        } else {
            uint32_t shmBytes = threadsY * VAR_MEM_PER_VERTEX_BYTES_DEFINE + (2 * hashArrayEntriesPerComm) * sizeof(KeyValueInt) * threadsY;
            assert(shmBytes <= SHARED_MEM_SIZE);

            reassign_huge_nodes<<<blockNum, dim, shmBytes>>> (binNodesNum, binNodes, 
                V, E, W, k, ac, comm, newComm, maxDegree, threadsY, hashArrayEntriesPerComm, m, nullptr);
        }
    }
    return 21.37;
}

__global__ 
void computeEiToCiSum(uint32_t V_MAX_IDX,
                        float*    __restrict__ ei_to_Ci,
                         const uint32_t* __restrict__ V,
                         const uint32_t* __restrict__ E,
                         const float*    __restrict__ W,
                         const uint32_t* __restrict__ comm) {
    uint32_t me = 1 + getGlobalIdx();

    if (me > V_MAX_IDX)
        return;

    uint32_t my_com = comm[me];

    for(uint32_t i = 0; i < V[me + 1] - V[me]; i++) {
        uint32_t comj = comm[ E[V[me] + i] ];

        if (my_com == comj) {
            atomicAdd(ei_to_Ci, W[V[me] + i]);
        }
    }
}

__host__ 
float __computeMod(float ei_to_Ci_sum, float m, const float* ac, uint32_t V_MAX_IDX) {
    auto tmp = thrust::device_vector<float>(V_MAX_IDX + 1);
    thrust::transform(ac, ac + V_MAX_IDX + 1, tmp.begin(), thrust::square<float>());
    float sum = thrust::reduce(tmp.begin(), tmp.end(), (double) 0, thrust::plus<double>());

    return ei_to_Ci_sum / (2 * m) - ( sum / (4 * m * m));
}


__host__
float computeModAndAC(uint32_t V_MAX_IDX,
                const uint32_t* __restrict__ V,
                const uint32_t* __restrict__ E,
                const float*    __restrict__ W,
                const float*    __restrict__ k,
                const uint32_t* __restrict__ comm,
                float* __restrict__ ac,
                float m) {
    
    float* ei_to_Ci;
    HANDLE_ERROR(hipHostAlloc((void**)&ei_to_Ci, sizeof(float), hipHostMallocDefault));
    *ei_to_Ci = 0;
    HANDLE_ERROR(hipHostGetDevicePointer(&ei_to_Ci, ei_to_Ci, 0));
    
    auto all_nodes_pair = getBlockThreadSplit(V_MAX_IDX);

    computeEiToCiSum <<<all_nodes_pair.first, all_nodes_pair.second>>> (V_MAX_IDX, ei_to_Ci, V, E, W, comm);
    hipDeviceSynchronize();

    zeroAC(ac, V_MAX_IDX);
    computeAC<<<all_nodes_pair.first, all_nodes_pair.second>>> (V_MAX_IDX, k, ac, comm);
    hipDeviceSynchronize();

    return __computeMod(*ei_to_Ci, m, ac, V_MAX_IDX);
}


__host__ 
float reassign_communities(
                        const uint32_t V_MAX_IDX,
                        uint32_t* __restrict__ V, 
                        uint32_t* __restrict__ E,
                        float*    __restrict__ W,
                        float*    __restrict__ k,
                        float*    __restrict__ ac,
                        uint32_t* __restrict__ comm,
                        uint32_t* __restrict__ newComm,
                        const float m,
                        const float minGain,
                        thrust::device_vector<uint32_t>& globCommAssignment) {

    uint32_t* binsHost = (uint32_t*) malloc(sizeof(BINS));
    hipMemcpyFromSymbol(binsHost, HIP_SYMBOL(BINS), sizeof(BINS), 0, hipMemcpyDeviceToHost);


    thrust::device_vector<uint32_t> G(V_MAX_IDX);
    thrust::sequence(G.begin(), G.end(), 1);

    // when running with --verbose option, we must keep proper community mapping
    // (community indeices are reassigned during contract phase) 
    thrust::sequence(globCommAssignment.begin(), globCommAssignment.end());

    auto partitionGenerator = [=](int rightIdx) {
        return [=] __device__ (const uint32_t& i) {
            return V[i + 1] - V[i] <= BINS[rightIdx];
        };
    };

    float mod0, mod1, maxMod;    
    mod0 = computeModAndAC(V_MAX_IDX, V, E, W, k, comm, ac, m);
    maxMod = mod0;

    bool changedSth = true;

    while(true) {

        // [0,1) is handled separately (lonely nodes; their modularity impact is 0)
        auto it0 = thrust::partition(G.begin(), G.end(), partitionGenerator(0));

        // for each bin sequentially computes new communities
        for (int i = 1; ; i++) {
            auto it = thrust::partition(it0, G.end(), partitionGenerator(i));
            uint32_t maxDegree = binsHost[i];
            
            uint32_t binNodesNum = thrust::distance(it0, it);
            if (binNodesNum == 0)
                break;

            uint32_t* binNodes = RAW(it0);

            reassign_communities_bin(binNodes, binNodesNum, V, E, W, k, ac, comm, newComm, maxDegree, m);

            hipDeviceSynchronize();

            auto pair = getBlockThreadSplit(binNodesNum);

            // update newComm table
            updateSpecific<<<pair.first, pair.second>>> (binNodes, binNodesNum, newComm, comm, V);
            hipDeviceSynchronize();

            // recompute AC values
            zeroAC(ac, V_MAX_IDX);
            computeAC<<<pair.first, pair.second>>> (V_MAX_IDX, k, ac, comm);
            hipDeviceSynchronize();

            it0 = it;
        }

        // OK, we computed new communities for all bins, let's check whether
        // modularity gain is satisfying

        mod1 = computeModAndAC(V_MAX_IDX, V, E, W, k, comm, ac, m);

        maxMod = max(maxMod, mod1);

        if (abs(mod1 - mod0) <= 0.001) {
            if (!changedSth) {
                return maxMod;
            } else {
                contract(V_MAX_IDX, V, E, W, k, comm, globCommAssignment);
                changedSth = false;
            }
        } else if (mod1 - mod0 < minGain) {
            contract(V_MAX_IDX, V, E, W, k, comm, globCommAssignment);
            changedSth = false;
            hipDeviceSynchronize();
            mod0 = mod1;
        } else {
            changedSth = true;
            mod0 = mod1;
        }
    }

    return mod1;
}

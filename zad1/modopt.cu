#include "hip/hip_runtime.h"
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <cassert>

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <set>
#include <float.h>

#include <thrust/device_vector.h>
#include <thrust/partition.h>
#include <thrust/sequence.h>
#include <thrust/distance.h>
#include <thrust/functional.h>

#include "hasharray.h"
#include "utils.h"

#include "modopt.h"


__host__
void contract(const uint32_t V_MAX_IDX,
                          uint32_t* __restrict__ V, 
                          uint32_t* __restrict__ E,
                          float*    __restrict__ W,
                          float*    __restrict__ k,
                          const uint32_t* __restrict__ comm,
                          thrust::device_vector<uint32_t>& globalCommAssignment);


using HA = HashArray;

// [ BINS[i], BINS[i+1] ) (right side exclusive)
__device__ uint32_t BINS[] =  
    {
        0, // [0,1) is handled separately (lonely nodes; their modularity impact is 0)
        4,
        8,
        16,
        32,
        96,
        320,
        1024,
        10000,
        20000,
        100000,
        UINT32_MAX // hash arrays in global memory
    };

__global__
void computeAC(const uint32_t V_MAX_IDX,
                const float* __restrict__ k,
               float*       __restrict__ ac,
               const uint32_t*    __restrict__ comm) {

    int tid = 1 + threadIdx.x + (blockIdx.x * blockDim.x);
    if (tid > V_MAX_IDX)
        return;
    atomicAdd(&ac[comm[tid]], k[tid]);
}

__host__
void zeroAC(float* ac, uint32_t V_MAX_IDX) {
    std::memset(ac, 0, sizeof(float) * (V_MAX_IDX + 1));
}

__device__ 
__forceinline__ 
unsigned int __lane_id() { 
    unsigned int laneid; 
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(laneid)); 
    return laneid; 
}



// https://fgiesen.wordpress.com/2013/01/21/order-preserving-bijections/
__device__
__forceinline__
uint32_t int_to_uint(int32_t val) {
    return val ^ 0x80000000;
}

__device__
__forceinline__
int32_t uint_to_int(uint32_t val) {
    return val ^ 0x80000000;
}

__device__
__forceinline__
int32_t float_to_int(float f32_val) {
    int32_t tmp = __float_as_int(f32_val);
    return tmp ^ ((tmp >> 31) & 0x7fffffff);
}

__device__
__forceinline__
float int_to_float(int32_t i32_val) {
    int32_t tmp = i32_val ^ ((i32_val >> 31) & 0x7fffffff);
    return __int_as_float(tmp);
}

__device__
__forceinline__
float uint_to_float(uint32_t ui32_val) {
    int32_t tmp1 = ui32_val ^ 0x80000000;
    int32_t tmp2 = tmp1 ^ (tmp1 >> 31);
    return __uint_as_float(tmp2);
}

__device__
__forceinline__
uint32_t float_to_uint(float f32_val) {
    uint32_t tmp = __float_as_uint(f32_val);
    return tmp ^ (((int32_t)tmp >> 31) | 0x80000000);
}

#define VAR_MEM_PER_VERTEX_BYTES_DEFINE 16


__device__ int numChanged = 0;

/**
 * Version of 'reassign_nodes' that handles nodes with degree > 32.
 * It deduces whether total size of hasharrays fits into shared memory,
 * if no, then it uses global memory.
 */
__global__ 
void reassign_huge_nodes(
                        const uint32_t  numNodes,
                        const uint32_t* __restrict__ binNodes,
                        const uint32_t* __restrict__ V,
                        const uint32_t* __restrict__ E,
                        const float*    __restrict__ W,
                        const float*    __restrict__ k,
                        const float*    __restrict__ ac,
                        const uint32_t* __restrict__ comm,
                        uint32_t*       __restrict__ newComm,
                        const uint32_t maxDegree,
                        const uint32_t nodesPerBlock,
                        const uint32_t hasharrayEntries, // @@@@@@@ >>>     TODO TODO TODO TODO TODO TODO <<<< @@@@@@@@
                        const float m,
                        const char*    globalHasharray,
                        const uint32_t stride,
                        const bool shared,
                        char *perVertexVars) {

    extern __shared__ char shared_mem[]; // shared memory is one-byte char type, for easy offset applying

    int i_ptr = threadIdx.y + blockIdx.x * blockDim.y; // my vertex pointer
    int edge_ptr = threadIdx.x; // my edge pointer

    if (i_ptr + edge_ptr == 0)
        numChanged = 0;
    
    assert(edge_ptr < 32);

    // before any early return, let's utilize all threads for zeroing memory.

    KeyValueFloat* hashWeight;
    KeyValueInt*   hashComm;

    // TODO: customize these
    // ei_to_Ci, loop, deltaModRes
    uint32_t lol = sizeof(int32_t) + sizeof(int32_t) + sizeof(uint64_t);
    assert(VAR_MEM_PER_VERTEX_BYTES_DEFINE == lol);

    // very careful pointer handling here, because of lots of type mismatches and casting
    if (shared) {
        uint32_t COMMON_VARS_SIZE_BYTES = VAR_MEM_PER_VERTEX_BYTES_DEFINE * nodesPerBlock;
        uint32_t off = COMMON_VARS_SIZE_BYTES + (i_ptr % nodesPerBlock) * (2 * hasharrayEntries) * sizeof(KeyValueFloat);
        hashWeight = (KeyValueFloat*) (&shared_mem[off]);
        assert( shared_mem + COMMON_VARS_SIZE_BYTES <= (char*) hashWeight);
        hashComm   = ((KeyValueInt*)   hashWeight) + hasharrayEntries;
        assert(globalHasharray == nullptr);
    } else {
        // printf("ENTRIES: %d, num: %d  \n", hasharrayEntries, i_ptr * (2 * hasharrayEntries));
        assert(globalHasharray != nullptr);
        assert(perVertexVars != nullptr);
        
        hashWeight = ( (KeyValueFloat*) globalHasharray ) + i_ptr * (2 * hasharrayEntries);
        hashComm   = ((KeyValueInt*) hashWeight) + hasharrayEntries;
        // printf("Kurwa: supp_diff: %d, %p, %p\n", hasharrayEntries * 8, (void*) hashWeight, (void*) hashComm);
    }

    {
        uint64_t* tmp = (uint64_t*) (shared ? shared_mem : perVertexVars);
        uint32_t perVertexVarsBytes = VAR_MEM_PER_VERTEX_BYTES_DEFINE * (shared ? nodesPerBlock : numNodes);
        for (int i = 0; i < 1 + perVertexVarsBytes / 8; i++) {
            tmp[i] = 0;
        }
    }

    // TODO za dużo roboty
    for (int i = edge_ptr; i < hasharrayEntries; i += 32) {
        hashWeight[i] = {.key = hashArrayNull, .value = (float) 0}; // 0 for easy atomicAdd
        hashComm[i]   = {.key = hashArrayNull, .value = hashArrayNull};
    }

    if (numNodes -1 < i_ptr) {
        // printf("node:%u  - nie istnieję, jestem narzutem na blok\n", i_ptr);
        return;
    }
    int i = binNodes[i_ptr]; // my vertex


    // variables common for each vertex, accumulating ei_to_Ci value 
    // computed in parallel
    char* realPerVertexVars = (shared ? shared_mem : perVertexVars);
    // if (i_ptr + edge_ptr == 0) {
    //     printf("NO LOL :  : :: !!!! ! %p\n", (void*)(realPerVertexVars + numNodes * VAR_MEM_PER_VERTEX_BYTES_DEFINE));
    // }
    uint32_t ei_to_ci_off_bytes = (i_ptr % (shared ? nodesPerBlock : numNodes)) * VAR_MEM_PER_VERTEX_BYTES_DEFINE;
    int32_t* glob_ei_to_Ci = (int32_t*) &realPerVertexVars[ei_to_ci_off_bytes];
    
    uint32_t loop_off_bytes = ei_to_ci_off_bytes + sizeof(int32_t);
    int32_t* glob_loop = (int32_t*) &realPerVertexVars[loop_off_bytes];

    uint32_t deltaMod_off_bytes = loop_off_bytes + sizeof(int32_t);
    uint64_t* glob_deltaMod = (uint64_t*) &realPerVertexVars[deltaMod_off_bytes];

    assert(*glob_ei_to_Ci == 0);
    assert(*glob_loop == 0);
    assert(*glob_deltaMod == 0);

        
    __syncthreads();

    uint64_t deltaMod;
    float ei_to_Ci;

    int cntr = 0;
    uint32_t edge_base = edge_ptr;
    uint32_t eitociidx = 0;

    while (true) {
        uint32_t EDGE = edge_base + cntr * stride;
        if (V[i + 1] - V[i] -1 < EDGE) {
            break;
        }

        uint32_t j = E[V[i] + EDGE];

        uint32_t mySlot = HA::insertInt(hashComm, comm[j], comm[j], hasharrayEntries);
        assert(hashComm[mySlot].key == comm[j]);
        HA::addFloatSpecificPos(hashWeight, mySlot, W[V[i] + EDGE]);

        if (comm[j] == comm[i])
            eitociidx = mySlot;

        cntr++;
    }

    __syncwarp();
    
    float eitocival = eitociidx == 0 ? 0.0 : hashWeight[eitociidx].value;

    for (int offset = 32 / 2; offset > 0; offset /= 2) {
        eitocival = max(eitocival, __shfl_down_sync(FULL_MASK, eitocival, offset));
    }

    assert(eitocival >= 0.0);

    float myBestGain = 0.0;
    uint32_t myBestComm = 0;
    cntr = 0;
    while (true) {
        uint32_t EDGE = edge_base + cntr * stride;
        if (V[i + 1] - V[i] -1 < EDGE) {
            break;
        }

        uint32_t j = E[V[i] + EDGE];

        uint32_t mySlot = HA::insertDummy(hashComm, comm[j], hasharrayEntries);
        float gain = k[i] * ( ac[comm[i]] - k[i] - ac[comm[j]] ) / (2 * m * m)  +  hashWeight[mySlot].value / m;
        gain += eitocival / m;

        if (gain <= 0.0) {
            cntr++;
            continue;
        }

        if (gain == myBestGain && comm[j] < myBestComm) {
            myBestComm = comm[j];
        } else if (gain > myBestGain) {
            myBestComm = comm[j];
            myBestGain = gain;
        }
        cntr++;
    }

    __syncwarp();

    float bestGainGlobal = 0.0;
    uint32_t bestCommGlobal = 0;
    for (int offset = 32 / 2; offset > 0; offset /= 2) {
        float bestGain = __shfl_down_sync(FULL_MASK, myBestGain, offset);
        float bestComm = __shfl_down_sync(FULL_MASK, myBestComm, offset);
            if (bestGain > bestGainGlobal) {
                bestGainGlobal = bestGain;
                bestCommGlobal = bestComm;
            }
    }

    if (edge_base == 0) {
        if (bestGainGlobal > 0.0) {
            assert(bestCommGlobal > 0);
            newComm[i] = bestCommGlobal;
        } else {
            newComm[i] = comm[i];
        }
    }
}

/**
 * This kernel function converts multiple nodes,
 * each node got it's own part of shared memory for node-common data, i.a hashArrays.
 */
__global__ 
void reassign_nodes(
                        const uint32_t  numNodes,
                        const uint32_t* __restrict__ binNodes,
                        const uint32_t* __restrict__ V,
                        const uint32_t* __restrict__ E,
                        const float*    __restrict__ W,
                        const float*    __restrict__ k,
                        const float*    __restrict__ ac,
                        const uint32_t* __restrict__ comm,
                        uint32_t*       __restrict__ newComm,
                        const uint32_t maxDegree,
                        const uint32_t nodesPerBlock,
                        const uint32_t hasharrayEntries,
                        const float m) {

    extern __shared__ KeyValueFloat hashtables[];

    assert(next_2_pow(maxDegree) == maxDegree); // maxDegree is power of 2
    
    int i_ptr =  threadIdx.y + blockIdx.x * blockDim.y; // threadIdx.y + (blockIdx.y * blockDim.y); // my node pointer
    int edgeNum = threadIdx.x; // my edge pointer

    if (numNodes -1 < i_ptr) {
        return;
    }

    int i = binNodes[i_ptr];

    if (V[i + 1] - V[i] -1 < edgeNum) {
        return;
    }

    uint32_t j = E[V[i] + edgeNum];

    // each hashtable contains of hasharrayEntries elements
    KeyValueFloat* hashWeight = (KeyValueFloat*) hashtables + (i_ptr % nodesPerBlock) * (2 * hasharrayEntries);
    KeyValueInt*   hashComm   = (KeyValueInt*)   hashWeight + hasharrayEntries;

    // TODO tu jest za dużo roboty
    for (int i = 0; i < hasharrayEntries; i++) {
        hashWeight[i].key = hashArrayNull;
        hashWeight[i].value = (float) 0;
        hashComm[i].key = hashArrayNull;
        hashComm[i].value = hashArrayNull;
    }
    __syncwarp();

    // ok, data initialized, let the run start
    uint32_t mySlot = HA::insertInt(hashComm, comm[j], comm[j], hasharrayEntries);
    float sum = HA::addFloatSpecificPos(hashWeight, mySlot, W[V[i] + edgeNum]);

    __syncwarp();

    uint32_t mask = __ballot_sync(FULL_MASK, edgeNum < maxDegree / 2);
    assert(mask == __ballot_sync(__activemask(), edgeNum < maxDegree / 2)); // TODO - to się kiedyś wysypie i dobrze, wtedy podmienić mask

    // sum of weights from node i to Ci\{i} from paper doesn't use loop values
    float loop = i == j ? W[V[i] + edgeNum] : 0;
    float ei_to_Ci = comm[j] == comm[i] ? hashWeight[mySlot].value : 0;
    float todo = i == j ? W[V[i] + edgeNum] : 0;

    __syncwarp();

    for (int offset = maxDegree / 2; offset > 0; offset /= 2) {
        ei_to_Ci = fmaxf(ei_to_Ci, __shfl_down_sync(mask, ei_to_Ci, offset)); // only warp with idx % maxDegree == 0 keeps proper value
        loop = fmaxf(loop, __shfl_down_sync(mask, loop, offset));
        todo += __shfl_down_sync(mask, todo, offset);
    }

    if (edgeNum == 0) {
        assert(todo == loop);
        ei_to_Ci -= loop;
    }
   

    // lack of -(e_i -> C_i\{i} / m) addend in that sum, it will be computed later
    float deltaMod = comm[j] >= comm[i] ? 
        -(1 << 5) : 
        k[i] * ( ac[comm[i]] - k[i] - ac[comm[j]] ) / (2 * m * m)  +  hashWeight[mySlot].value / m;

    uint32_t newCommIdx = comm[j];

    for (int offset = maxDegree / 2; offset > 0; offset /= 2) {
        float deltaModRed = __shfl_down_sync(mask, deltaMod, offset);
        uint32_t newCommIdxRed = __shfl_down_sync(mask, newCommIdx, offset);

        if (newCommIdxRed == 0)
            continue; // TODO undefined behavior

        if (deltaModRed > deltaMod) {
            deltaMod = deltaModRed;
            newCommIdx = newCommIdxRed;
        } else if (deltaModRed == deltaMod) {
            newCommIdx = (uint32_t) fminf((uint32_t) newCommIdx, (uint32_t) newCommIdxRed);
        }
    }

    if (edgeNum == 0) {
        float gain = deltaMod - ei_to_Ci / m;

        if (gain > 0 && newCommIdx < comm[i]) {
            assert(newCommIdx != 0);
            newComm[i] = newCommIdx;
        } else {
            newComm[i] = comm[i];
        }
    }
}

__host__ 
float reassign_communities_bin(
                        const uint32_t* binNodes,
                        const uint32_t binNodesNum,
                        const uint32_t* __restrict__ V, 
                        const uint32_t* __restrict__ E,
                        const float*    __restrict__ W,
                        const float*    __restrict__ k,
                        const float*    __restrict__ ac,
                        uint32_t* __restrict__ comm,
                        uint32_t* __restrict__ newComm,
                        uint32_t maxDegree,
                        const float m) {

    // TODO customize this, maybe check 2 * maxDegree?
    uint32_t hashArrayEntriesPerComm = next_2_pow(maxDegree + 1);

    int stride = 32;
    uint32_t threadsX = min(maxDegree, 32);
    
    uint32_t maxThreadsY = 512 / threadsX; // TODO customize

    uint32_t threadsY = min(maxThreadsY, binNodesNum);
    uint32_t blockNum = ceil( (float)binNodesNum / (float) threadsY );

    dim3 dim(threadsX, threadsY);

    if (maxDegree > 32) {
        // huge nodes, maybe that huge that hasharrays cannot fit in shared mem

        char *deviceGlobalHashArrays = nullptr, *perVertexVars = nullptr;

        // it will be increased conditionally later
        uint32_t shmBytes = threadsY * VAR_MEM_PER_VERTEX_BYTES_DEFINE;
        assert(shmBytes <= SHARED_MEM_SIZE);

        bool useGlobalMem = SHARED_MEM_SIZE < shmBytes + (2 * hashArrayEntriesPerComm) * sizeof(KeyValueInt) * threadsY;

        if (useGlobalMem) {
            size_t memsize = sizeof(KeyValueFloat) * binNodesNum * (2 * hashArrayEntriesPerComm);
            HANDLE_ERROR(hipMalloc(&deviceGlobalHashArrays, 2 * memsize));

            size_t perVertexMemSize = VAR_MEM_PER_VERTEX_BYTES_DEFINE * binNodesNum;
            HANDLE_ERROR(hipMalloc(&perVertexVars, 2 * perVertexMemSize));
            assert(perVertexVars != nullptr);
        } else {
            shmBytes += (2 * hashArrayEntriesPerComm) * sizeof(KeyValueInt) * threadsY;
            assert(shmBytes <= SHARED_MEM_SIZE);            
        }

        reassign_huge_nodes<<<blockNum, dim, shmBytes>>> (binNodesNum, binNodes, 
            V, E, W, k, ac, comm, newComm, maxDegree, threadsY, hashArrayEntriesPerComm, m, deviceGlobalHashArrays, stride, !useGlobalMem, perVertexVars);

        hipDeviceSynchronize();

        if (useGlobalMem) {
            assert(deviceGlobalHashArrays != nullptr);
            assert(perVertexVars != nullptr);
            
            HANDLE_ERROR(hipFree(deviceGlobalHashArrays));
            HANDLE_ERROR(hipFree(perVertexVars));
        }

    } else {
        uint32_t shmBytes = (2 * hashArrayEntriesPerComm) * sizeof(KeyValueInt) * threadsY;
        assert(shmBytes <= SHARED_MEM_SIZE);

        reassign_nodes<<<blockNum, dim, shmBytes>>>      (binNodesNum, binNodes, 
            V, E, W, k, ac, comm, newComm, maxDegree, threadsY, hashArrayEntriesPerComm, m);
    }

    return 21.37;
}

__global__ 
void computeEiToCiSum(uint32_t V_MAX_IDX,
                        float*    __restrict__ ei_to_Ci,
                         const uint32_t* __restrict__ V,
                         const uint32_t* __restrict__ E,
                         const float*    __restrict__ W,
                         const uint32_t* __restrict__ comm) {
    uint32_t me = 1 + getGlobalIdx();

    if (me > V_MAX_IDX)
        return;

    uint32_t my_com = comm[me];

    for(uint32_t i = 0; i < V[me + 1] - V[me]; i++) {
        uint32_t comj = comm[ E[V[me] + i] ];

        if (my_com == comj) {
            atomicAdd(ei_to_Ci, W[V[me] + i]);
        }
    }
}

__host__ 
float __computeMod(float ei_to_Ci_sum, float m, const float* ac, uint32_t V_MAX_IDX) {
    auto tmp = thrust::device_vector<float>(V_MAX_IDX + 1);
    thrust::transform(ac, ac + V_MAX_IDX + 1, tmp.begin(), thrust::square<float>());
    float sum = thrust::reduce(tmp.begin(), tmp.end(), (double) 0, thrust::plus<double>());
    return ei_to_Ci_sum / (2 * m) - ( sum / (4 * m * m));
}


__host__
float computeModAndAC(uint32_t V_MAX_IDX,
                const uint32_t* __restrict__ V,
                const uint32_t* __restrict__ E,
                const float*    __restrict__ W,
                const float*    __restrict__ k,
                const uint32_t* __restrict__ comm,
                float* __restrict__ ac,
                float m) {
    
    float* ei_to_Ci;
    HANDLE_ERROR(hipHostAlloc((void**)&ei_to_Ci, sizeof(float), hipHostMallocDefault));
    *ei_to_Ci = 0;
    HANDLE_ERROR(hipHostGetDevicePointer(&ei_to_Ci, ei_to_Ci, 0));
    
    auto all_nodes_pair = getBlockThreadSplit(V_MAX_IDX);

    computeEiToCiSum <<<all_nodes_pair.first, all_nodes_pair.second>>> (V_MAX_IDX, ei_to_Ci, V, E, W, comm);
    hipDeviceSynchronize();

    zeroAC(ac, V_MAX_IDX);
    computeAC<<<all_nodes_pair.first, all_nodes_pair.second>>> (V_MAX_IDX, k, ac, comm);
    hipDeviceSynchronize();

    return __computeMod(*ei_to_Ci, m, ac, V_MAX_IDX);
}


__host__ 
float reassign_communities(
                        const uint32_t V_MAX_IDX,
                        uint32_t* __restrict__ V, 
                        uint32_t* __restrict__ E,
                        float*    __restrict__ W,
                        float*    __restrict__ k,
                        float*    __restrict__ ac,
                        uint32_t* __restrict__ comm,
                        uint32_t* __restrict__ newComm,
                        const float m,
                        const float minGain,
                        thrust::device_vector<uint32_t>& globCommAssignment) {

    // TODO free
    uint32_t* binsHost = (uint32_t*) malloc(sizeof(BINS));
    hipMemcpyFromSymbol(binsHost, HIP_SYMBOL(BINS), sizeof(BINS), 0, hipMemcpyDeviceToHost);


    thrust::device_vector<uint32_t> G(V_MAX_IDX);
    thrust::sequence(G.begin(), G.end(), 1);

    // when running with --verbose option, we must keep proper community mapping
    // (community indeices are reassigned during contract phase) 
    thrust::sequence(globCommAssignment.begin(), globCommAssignment.end());

    auto partitionGenerator = [=](int rightIdx) {
        return [=] __device__ (const uint32_t& i) {
            return V[i + 1] - V[i] <= BINS[rightIdx];
        };
    };

    float mod0, mod1, maxMod;    
    mod0 = computeModAndAC(V_MAX_IDX, V, E, W, k, comm, ac, m);
    maxMod = mod0;

    bool changedSth = true;

    while(true) {

        // [0,1) is handled separately (lonely nodes; their modularity impact is 0)
        auto it0 = thrust::partition(G.begin(), G.end(), partitionGenerator(0));

        // for each bin sequentially computes new communities
        for (int i = 1; ; i++) {
            if (i >= 11)
                break;
            auto it = thrust::partition(it0, G.end(), partitionGenerator(i));
            uint32_t maxDegree = binsHost[i];
            
            uint32_t binNodesNum = thrust::distance(it0, it);
            if (binNodesNum == 0) {
                assert(it0 == it);
                continue;
            }

            uint32_t* binNodes = RAW(it0);

            reassign_communities_bin(binNodes, binNodesNum, V, E, W, k, ac, comm, newComm, maxDegree, m);

            hipDeviceSynchronize();

            auto pair = getBlockThreadSplit(binNodesNum);

            // update newComm table
            updateSpecific<<<pair.first, pair.second>>> (binNodes, binNodesNum, newComm, comm, V);
            hipDeviceSynchronize();

            // recompute AC values
            computeModAndAC(V_MAX_IDX, V, E, W, k, comm, ac, m);
            hipDeviceSynchronize();

            it0 = it;
        }

        hipDeviceSynchronize();

        mod1 = computeModAndAC(V_MAX_IDX, V, E, W, k, comm, ac, m);

        maxMod = max(maxMod, mod1);

        if (abs(mod1 - mod0) <= 0.001) {
            if (!changedSth) {
                return maxMod;
            } else {
                contract(V_MAX_IDX, V, E, W, k, comm, globCommAssignment);
                changedSth = false;
                float mod_contract = computeModAndAC(V_MAX_IDX, V, E, W, k, comm, ac, m);
            }
        } else if (mod1 - mod0 < minGain) {
            contract(V_MAX_IDX, V, E, W, k, comm, globCommAssignment);
            changedSth = false;
            hipDeviceSynchronize();
            mod0 = mod1;

            float mod_contract = computeModAndAC(V_MAX_IDX, V, E, W, k, comm, ac, m);
        } else {
            changedSth = true;
        }
        mod0 = mod1;
    }

    return mod1;
}

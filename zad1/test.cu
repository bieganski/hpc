#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/partition.h>
#include <iostream>

#include "hasharray.h"

// using namespace std;

typedef HashArray HA;

// struct pred {
//     __host__ __device__
//     bool operator()(const int &x) {
//         return x < 3;
//     }
// };

__global__ void kernel(float* ptr) { //KeyValueFloat* hashtable

    extern __shared__ float arr[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    arr[490000 * 1024 / sizeof(float)] = 0.2;

    *ptr = arr[490000 * 1024 / sizeof(float) - 15 + tid];

    if (tid > 30)
        return;

    __syncwarp((1 << 5) - 1);

    // if(tid == 0){
    //     ;
    // }
    // else {
    //     return;
    // }
        
    // __syncthreads();
    
    // uint32_t res_key = HA::addFloat(hashtable, 1, 1.01, 2 << 5);

    // printf("%d: wstawilem pod %d, patrze: %f\n", tid, res_key, hashtable[res_key].value);
}


static void HandleError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__ ))


// __global__ void kernel() {
//   __shared__ int semaphore;
//   semaphore=0;
//   __syncthreads();
//   while (true) {
//     int prev=atomicCAS(&semaphore,0,1);
//     if (prev==0) {
//       //critical section
//       semaphore=0;
//       break;
//     }
//   }
// }

__device__ uint32_t CONTRACT_BINS[] = {
    0,
    121,
    385,
    UINT32_MAX
};

int main(void)
{
    KeyValueFloat* hashtable;

    HANDLE_ERROR(hipMalloc((void**) &hashtable, sizeof(KeyValueFloat) * (2 << 5)));
    hipDeviceSynchronize();

    uint32_t* ptr = (uint32_t*) malloc(4 * 6);

    hipMemcpyFromSymbol(ptr, HIP_SYMBOL(CONTRACT_BINS), 4 * 4, 0, hipMemcpyDeviceToHost);

    // hipDeviceSynchronize();
    // HA::init(hashtable, 2 << 5);

    printf("LOL: %d\n", ptr[2]);
    printf("LOLSIZE: %d\n", sizeof(CONTRACT_BINS));

    // kernel<<<1, 16, 490000 * 1024>>>((float*) hashtable);

    hipDeviceSynchronize();
    return 0;
}














    // initialize all ten integers of a device_vector to 1
    // thrust::device_vector<int> D(10, 1);

    // // set the first seven elements of a vector to 9
    // thrust::fill(D.begin(), D.begin() + 7, 9);

    // thrust::sequence(D.begin(), D.end());

    // // print D
    // for(int i = 0; i < D.size(); i++)
    //     std::cout << "D[" << i << "] = " << D[i] << std::endl;

    // struct pred p;
    // auto it = thrust::partition(D.begin(), D.end(), p);


    // for(int i = 0; i < D.size(); i++)
    //     std::cout << "D[" << i << "] = " << D[i] << std::endl;

    // cout << endl << *it;
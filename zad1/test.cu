#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/partition.h>
#include <iostream>

#include "hasharray.h"

// using namespace std;

typedef HashArray HA;


static void HandleError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__ ))
#define FULL_MASK 0xFFFFFFFF


// struct pred {
//     __host__ __device__
//     bool operator()(const int &x) {
//         return x < 3;
//     }
// };

#include <float.h>


__device__ 
void binprintf(uint32_t v)
{
    uint32_t mask = 1 << ((sizeof(uint32_t) << 3) - 1);
    while (mask) {
        printf("%u", (v & mask ? 1 : 0));
        mask >>= 1;
    }
    printf("\n");
}
__device__ __forceinline__ unsigned int __laneid() { unsigned int laneid; asm volatile ("mov.u32 %0, %laneid;" : "=r"(laneid)); return laneid; }

__global__ 
void wtf(uint32_t* ptr) { //KeyValueFloat* hashtable

    extern __shared__ float arr[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Tid: %d\n", __laneid());

    if (tid != 0)
        return;
    printf("%f\n", -FLT_MAX);

    printf("%f\n", -FLT_MAX + FLT_MAX);

    printf("%f\n", -FLT_MAX + 1000);

    return;


    int val = tid;
    if (tid == 1)
        return;

    int mask = 0x00000003; // FULL_MASK; // __activemask(); // __ballot_sync(FULL_MASK, 1);
    
    for (int offset = 2; offset > 0; offset /= 2) {
        val = fminf(val, __shfl_down_sync(mask, val, offset)); // only warp with idx == 0 keeps proper value
        if (tid == 0) {
            printf("_%d_\n", val);
        }
    }
        


    // int leader = __ffs(mask) - 1;
    // if (tid == 0) {
    //     printf("leader = %d\n", leader);
    //     printf("mask:\n");
    //     binprintf(mask);
    // }
    // int val = __laneid() == leader ? 5 : 1;

    // int res = __shfl_sync(mask, val, leader);

    if (tid == 0)
        printf("val = %d\n", val);



    // if(tid == 0){
    //     ;
    // }
    // else {
    //     return;
    // }
        
    // __syncthreads();
    
    // uint32_t res_key = HA::addFloat(hashtable, 1, 1.01, 2 << 5);

    // printf("%d: wstawilem pod %d, patrze: %f\n", tid, res_key, hashtable[res_key].value);
}


// __global__ void kernel() {
//   __shared__ int semaphore;
//   semaphore=0;
//   __syncthreads();
//   while (true) {
//     int prev=atomicCAS(&semaphore,0,1);
//     if (prev==0) {
//       //critical section
//       semaphore=0;
//       break;
//     }
//   }
// }

__device__ uint32_t CONTRACT_BINS[] = {
    0,
    121,
    385,
    UINT32_MAX
};

int main(void)
{
    KeyValueFloat* hashtable;

    HANDLE_ERROR(hipHostAlloc((void**)&hashtable, sizeof(KeyValueFloat) * (2 << 5), hipHostMallocDefault));
    hipDeviceSynchronize();

    // uint32_t* ptr = (uint32_t*) malloc(4 * 6);

    // hipMemcpyFromSymbol(ptr, HIP_SYMBOL(CONTRACT_BINS), 4 * 4, 0, hipMemcpyDeviceToHost);

    // hipDeviceSynchronize();
    // HA::init(hashtable, 2 << 5);

    // printf("LOL: %d\n", ptr[2]);
    // printf("LOLSIZE: %d\n", sizeof(CONTRACT_BINS));

    wtf<<<1, 32, 48 * 1024>>>((uint32_t*) hashtable);

    hipDeviceSynchronize();

    printf("WYNIK: %d", * ((int*) hashtable));
    return 0;
}

#include "hip/hip_runtime.h"
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <cassert>
#include <math.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <set>

#include <thrust/device_vector.h>
#include <thrust/partition.h>
#include <thrust/sequence.h>
#include <thrust/distance.h>
#include <thrust/functional.h>

#include "hasharray.h"
#include "utils.h"
#include "modopt.h"

using namespace std; // TODO wywalić

using HA = HashArray;

float MIN_GAIN;
char *FILE_PATH;
bool VERBOSE = 0;

uint32_t* V; // vertices
uint32_t* E; // edges
float* W; // weights
float* k; // sum of weights per node
float* ac; // sum of weights per community
uint32_t* comm; // communities
uint32_t* newComm; // new communities after reassign phase
float m; // total sum of weights

uint32_t V_MAX_IDX;



__global__
void compute_size_degree(const uint32_t V_MAX_IDX,
                          const uint32_t* __restrict__ V,
                          const uint32_t* __restrict__ comm,
                          uint32_t* __restrict__ commSize,
                          uint32_t* __restrict__ commDegree) {
    int tid = 1 + getGlobalIdx();
    assert(tid <= V_MAX_IDX);

    atomicAdd(&commSize[comm[tid]], 1);
    atomicAdd(&commDegree[comm[tid]], V[tid + 1] - V[tid]);

    __syncthreads(); // TODO - wywalić

    if (tid == 1) {
        printf("WYPISUJĘ OBLICZONE COMMUNITY SIZES: \n");
        for (int i = 0; i <=V_MAX_IDX; i++) {
            printf("%d ", commSize[i]); 
        }
        printf("\n");


        printf("WYPISUJĘ OBLICZONE COMMUNITY DEGREES: \n");
        for (int i = 0; i <=V_MAX_IDX; i++) {
            printf("%d ", commDegree[i]); 
        }
        printf("\n");
    }
}


#define NODE_EXISTS(i, V, E) (V[i+1] - V[i] > 0)

__global__
void compute_compressed_comm(const uint32_t V_MAX_IDX,
                          const uint32_t* __restrict__ V,
                          const uint32_t* __restrict__ E,
                          const uint32_t* __restrict__ comm,
                          uint32_t* __restrict__ commSize,
                          uint32_t* __restrict__ vertexStart,
                          uint32_t* __restrict__ tmpCounter,
                          uint32_t* __restrict__ compressedComm) {
    int tid = 1 + getGlobalIdx();
    assert(tid <= V_MAX_IDX);

    if (!NODE_EXISTS(tid, V, E))
        return;

    int my_comm = comm[tid];

    int idx = atomicAdd(&tmpCounter[my_comm], 1);

    compressedComm[vertexStart[my_comm] + idx] = tid;
}

__device__ uint32_t CONTRACT_BINS[] = {
    0,
    16,
    384,
    UINT32_MAX
};

__host__
void computeWTF(const uint32_t* __restrict__ V,
                          thrust::device_vector<uint32_t>& compressedComm,
                          thrust::device_vector<uint32_t>& WTF) {
    thrust::transform(compressedComm.begin(), compressedComm.end(), WTF.begin(), 
        [=] __device__ (const uint32_t& i) {
            return V[i + 1] - V[i];
        });
    thrust::exclusive_scan(WTF.begin(), WTF.end(), WTF.begin());
}


__global__
void compute_comm_neighbors(
    const uint32_t* __restrict__ V,
    const uint32_t* __restrict__ E,
    const float*    __restrict__ W,
    const uint32_t* __restrict__ comm,
    const uint32_t* __restrict__ binCommunities,
    const uint32_t* __restrict__ vertexStart,
    const uint32_t* __restrict__ compressedComm,
    const uint32_t* __restrict__ edgePos,
    const uint32_t* __restrict__ WTF,
    const uint32_t hasharrayEntries,
    uint32_t* __restrict__ newV,
    uint32_t* __restrict__ newE,
    float*    __restrict__ newW,
    uint32_t* __restrict__ freeIndices
) {
    
    extern __shared__ KeyValueFloat hashtables[];

    int myCommPtr = blockIdx.x; // threadIdx.y + (blockIdx.y * blockDim.y);
    int myEdgePtr = threadIdx.x; //  + (blockIdx.x * blockDim.x);

    uint32_t myComm = binCommunities[myCommPtr];

    uint32_t firstNodePtrIncl = vertexStart[myComm];
    uint32_t lastNodePtrExcl  = vertexStart[myComm + 1];

    KeyValueFloat* hashWeight = (KeyValueFloat*) hashtables; // + myCommPtr * (2 * hasharrayEntries);
    KeyValueInt*   hashComm   = (KeyValueInt*)   hashWeight + hasharrayEntries;

    for (int i = myEdgePtr; i < hasharrayEntries; i += WARP_SIZE) {
        hashWeight[i] = {.key = hashArrayNull, .value = (float) 0}; // 0 for easy atomicAdd
        // hashWeight[i].key = hashArrayNull;
        // hashWeight[i].value = (float) 0;
        hashComm[i]   = {.key = hashArrayNull, .value = hashArrayNull};
        // hashComm[i].key = hashArrayNull;
        // hashComm[i].value = hashArrayNull;
    }

    __syncwarp();

    uint32_t insertedByMe = 0;
    uint32_t start = firstNodePtrIncl;
    uint32_t offset = WTF[firstNodePtrIncl]; // TODO benchmark bez tego

    bool finish = false; // cannot use early return because of usage of warp-level primitives
    while(true) {

        // looking for my node and edge
        uint32_t myEdge = -1;
        uint32_t myNode = -1;
        uint32_t edgeIdx = -1;

        // printf("debug: %d, %d\n", start, lastNodePtrExcl);
        // if (start >= lastNodePtrExcl)
        //     return ;
        for (uint32_t i = start; !finish && (i < lastNodePtrExcl); i++) {
            if (myEdgePtr < WTF[i + 1] - offset) {
                myNode = compressedComm[i];
                start = i; // for next iteration
                edgeIdx = myEdgePtr - (WTF[i] - offset);
                myEdge = E[V[myNode] + edgeIdx];
                printf("%d: dla myPtrEdge: %d znalazlem edge %d (edgeIdx = %d)\n", myNode, myEdgePtr, myEdge, edgeIdx);
                break;
            } else if (i == lastNodePtrExcl - 1) {
                // they don't need me :(
                // printf("wychodze bo mnie nie potrzebują: %d, %d\n", myComm, myEdgePtr);
                finish = true;
            }
        }

        if (finish)
            break;

        // I know who am I, now add my neighbor to sum of weights
        printf( "%d->%d: dodaje do haszarray wage %f, entries: %d\n", myNode, myEdge, W[V[myNode] + edgeIdx], hasharrayEntries);
        if ( HA::insertWithFeedback(hashComm, hashWeight, comm[myEdge], comm[myEdge], W[V[myNode] + edgeIdx], hasharrayEntries) ) {
            insertedByMe++;
        } else {
            printf("ooops! mamy konflikt!\n");
        }

        myEdgePtr += WARP_SIZE;
        printf("%d: inserted by me: %d\n", myEdgePtr % WARP_SIZE, insertedByMe);
    } // while(true)

    // now, compute number of totally inserted in this warp's community

    int mask = __activemask();

    // binprintf(mask);

    assert(mask == FULL_MASK);

    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
        insertedByMe += __shfl_down_sync(mask, insertedByMe, offset); // only warp with idx == 0 keeps proper value

    
    assert(0 == __ffs(mask) - 1);
    int leader = __ffs(mask) - 1; // = 0 // TODO assumption: zero-idx-thread is alive

    uint32_t commNeighborsNum = __shfl_sync(mask, insertedByMe, leader);

    int myEdgePtr0 = threadIdx.x;
    if (myEdgePtr0 == 0) {
        // WARNING: we use old community id, because we already know free E indices!
        newV[myComm] = commNeighborsNum; // will be computed prefix sum on it later
    }

    assert(mask == FULL_MASK); // !!!!!
    uint32_t idx0 = edgePos[myComm];
    // if (myEdgePtr0 == 0) {
    //     printf("%d: our idx0: %d\n", myComm, idx0);
    // }
    for (int i = myEdgePtr0; i < hasharrayEntries; i += WARP_SIZE) {
        if (hashComm[i].key != hashArrayNull) {
            uint32_t myIdx = atomicAdd(&freeIndices[myComm], 1);
            newE[idx0 + myIdx] = hashComm[i].value;
            newW[idx0 + myIdx] = hashWeight[i].value;
            // printf("%d: dodaje sasiada %d pod idx %d\n", myComm, hashComm[i].value, idx0 + myIdx); 
        }
    }
}


__host__
void contract(const uint32_t V_MAX_IDX,
                          const uint32_t* __restrict__ V, 
                          const uint32_t* __restrict__ E,
                          const float*    __restrict__ W,
                          const float*    __restrict__ k,
                          const uint32_t* __restrict__ comm) {

    // TODO przenieśc je wyżej, żeby alokować tylko raz
    thrust::device_vector<uint32_t> commSize(V_MAX_IDX + 1, 0);
    thrust::device_vector<uint32_t> commDegree(V_MAX_IDX + 1, 0);
    thrust::device_vector<uint32_t> edgePos(V_MAX_IDX + 1, 0);
    thrust::device_vector<uint32_t> newID(V_MAX_IDX + 1, 0);
    thrust::device_vector<uint32_t> vertexStart(V_MAX_IDX + 2, 0);
    thrust::device_vector<uint32_t> tmpCounter(V_MAX_IDX + 1, 0);
    thrust::device_vector<uint32_t> compressedComm(V_MAX_IDX + 1, 0);
    thrust::device_vector<uint32_t> commSeq(V_MAX_IDX + 1, 0);
    thrust::device_vector<uint32_t> freeIndices(V_MAX_IDX + 1, 0);

    thrust::device_vector<uint32_t> WTF(V_MAX_IDX + 1 , 0);

    thrust::sequence(commSeq.begin(), commSeq.end());

    auto pair = getBlockThreadSplit(V_MAX_IDX);
    compute_size_degree<<<pair.first, pair.second>>> (V_MAX_IDX, V, comm, RAW(commSize), RAW(commDegree));

    hipDeviceSynchronize();

    thrust::transform(commSize.begin(), commSize.end(), newID.begin(), [] __device__ (const uint32_t& size) {return size > 0 ? 1 : 0;});
    thrust::inclusive_scan(newID.begin(), newID.end(), newID.begin());
    
    thrust::inclusive_scan(commSize.begin(), commSize.end(), &vertexStart[1]); // start output at 1 
    printf("newID: \n");
    // PRINT(newID.begin(), newID.end());
    thrust::copy(newID.begin(),newID.end(), std::ostream_iterator<uint32_t>(std::cout, " "));
    printf("\n");

    compute_compressed_comm <<<pair.first, pair.second>>> (V_MAX_IDX, V, E, comm, 
            RAW(commSize), RAW(vertexStart), RAW(tmpCounter), RAW(compressedComm));


    hipDeviceSynchronize();

    printf("VERTEX START: \n");
    thrust::copy(vertexStart.begin(), vertexStart.end(), std::ostream_iterator<uint32_t>(std::cout, " "));


    printf("\nCOMPRESSED COMM: \n");
    thrust::copy(compressedComm.begin(), compressedComm.end(), std::ostream_iterator<uint32_t>(std::cout, " "));

    thrust::exclusive_scan(commDegree.begin(), commDegree.end(), edgePos.begin());

    printf("EDGE POS: \n");
    thrust::copy(edgePos.begin(), edgePos.end(), std::ostream_iterator<uint32_t>(std::cout, " "));


    auto commDegreeLambda = RAW(commDegree); // you cannot use thrust's vector in device code

    auto partitionGenerator = [=](int rightIdx) {
        return [=] __device__ (const uint32_t& i) {
            return commDegreeLambda[i] <= CONTRACT_BINS[rightIdx];
        };
    };

    // TODO to też powinno być na zewnątrz
    // TODO free
    uint32_t* contractBinsHost = (uint32_t*) malloc(sizeof(CONTRACT_BINS));
    hipMemcpyFromSymbol(contractBinsHost, HIP_SYMBOL(CONTRACT_BINS), sizeof(CONTRACT_BINS), 0, hipMemcpyDeviceToHost);

    computeWTF(V, compressedComm, WTF);

    printf("WTF: \n");
    // PRINT(WTF.begin(), WTF.end());
    thrust::copy(WTF.begin(), WTF.end(), std::ostream_iterator<uint32_t>(std::cout, " "));
    printf("\n");

    // we don't want empty communities
    auto it0 = thrust::partition(commSeq.begin(), commSeq.end(), partitionGenerator(0));
    auto it = it0;
    for (int i = 1; ; i++) {

        if (it0 == commSeq.end()) {
            break;
        }
        
        it = thrust::partition(it0, commSeq.end(), partitionGenerator(i));

        // ok, let's handle communities with same degree boundary
        uint32_t degUpperBound = contractBinsHost[i];
        printf("degUPPERbound: %d\n", degUpperBound);

        uint32_t totalNumCommunities = thrust::distance(it0, it);
        printf("Num comm: %d\n", totalNumCommunities);

        uint32_t hashArrayEntriesPerComm = degUpperBound; // TODO customize this
        // uint32_t hashArrayEntriesPerBlock = SHARED_MEM_SIZE / sizeof(KeyValueInt); // should be 6144
        // uint32_t maxNumCommunitiesPerBlock = hashArrayEntriesPerBlock / (2 * hashArrayEntriesPerComm);

        // printf("MAX COMM PER BLOCK: %d\n", maxNumCommunitiesPerBlock);
        // uint32_t numCommunities = std::min(totalNumCommunities, maxNumCommunitiesPerBlock);
        
        // dim3 dimBlock(degUpperBound, numCommunities); // x per edges, y per nodes
        // uint16_t blockNum = ceil((float) totalNumCommunities / (float) maxNumCommunitiesPerBlock);
        // printf("DIMS: %d, %d, BLOKI: %d\n", degUpperBound, numCommunities, blockNum);
        // TODO shm zeroing

        // compute_comm_neighbors <<<blockNum, dimBlock, SHARED_MEM_SIZE>>> (V, E, W, comm, RAW(it0),
        //     RAW(vertexStart), RAW(compressedComm), RAW(commDegree), RAW(WTF), hashArrayEntriesPerComm);

        // printf("config: %d, %d, %d\n", totalNumCommunities, WARP_SIZE, sizeof(KeyValueInt) * (2 * hashArrayEntriesPerComm));
        uint32_t E_size = V[V_MAX_IDX + 1];
        printf("E size: %d\n", E_size);
        thrust::device_vector<uint32_t> newV(V_MAX_IDX + 2, 0);
        thrust::device_vector<uint32_t> newE(E_size, 0);
        thrust::device_vector<float>    newW(E_size, 0);

        compute_comm_neighbors <<< totalNumCommunities, WARP_SIZE, sizeof(KeyValueInt) * (2 * hashArrayEntriesPerComm) >>> 
            (
                V, E, W, comm, 
                RAW(it0),
                RAW(vertexStart), 
                RAW(compressedComm), 
                RAW(edgePos), 
                RAW(WTF), 
                hashArrayEntriesPerComm, 
                RAW(newV), 
                RAW(newE), 
                RAW(newW), 
                RAW(freeIndices)
            );


        hipDeviceSynchronize();

        

        printf("\newID:");
        thrust::copy(newID.begin(), newID.end(), 
            std::ostream_iterator<uint32_t>(std::cout, " "));
        printf("\nnowe V:");
        thrust::copy(newV.begin(), newV.end(), 
            std::ostream_iterator<uint32_t>(std::cout, " "));
        printf("\nnowe E:");
        thrust::copy(newE.begin(), newE.end(), 
            std::ostream_iterator<uint32_t>(std::cout, " "));
        printf("\nnowe W:");
        thrust::copy(newW.begin(), newW.end(), 
            std::ostream_iterator<float>(std::cout, " "));

        thrust::device_vector<uint32_t> realNewE(vertexStart.size() - 1);
        thrust::device_vector<uint32_t> realNewW(newW.size());

        thrust::copy_if(newE.begin(), newE.end(), realNewE.begin(), [] __device__ (const uint32_t& x) {return x != 0;});
        thrust::copy_if(newW.begin(), newW.end(), realNewW.begin(), [] __device__ (const uint32_t& x) {return x != 0;});


        printf("\n prawdziwe E:");
        thrust::copy(realNewE.begin(), realNewE.end(), 
            std::ostream_iterator<float>(std::cout, " "));
        printf("\n prawdziwe W:");
        thrust::copy(realNewW.begin(), realNewW.end(), 
            std::ostream_iterator<float>(std::cout, " "));

        it0 = it; // it0 points to first node that wasn't processed yet
    }
    return;
}


/**
 * UWAGA:
 * N - rozmiar tablicy k
 * N + 1 - rozmiar tablicy V
 * V - iterujemy od 1
 * E, W - rozmiar 2*N, iterujemy od 0
 * */
int main(int argc, char **argv) {
    if (parse_args(argc, argv)) {
        exit(1);
    }
    
    ret_t res = parse_inut_graph(get_input_content());

    V_MAX_IDX = std::get<0>(res);
    m = std::get<5>(res);

    printf("loool\n");

    HANDLE_ERROR(hipHostGetDevicePointer(&V, std::get<1>(res), 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&E, std::get<2>(res), 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&W, std::get<3>(res), 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&k, std::get<4>(res), 0));

    uint32_t* tmp;
    HANDLE_ERROR(hipHostAlloc((void**)&tmp, sizeof(uint32_t) * (V_MAX_IDX + 1), hipHostMallocDefault));
    HANDLE_ERROR(hipHostGetDevicePointer(&newComm, tmp, 0));
    HANDLE_ERROR(hipHostAlloc((void**)&tmp, sizeof(float) * (V_MAX_IDX + 1), hipHostMallocDefault));
    HANDLE_ERROR(hipHostGetDevicePointer(&ac, tmp, 0));

    // communities separately, because they must be initialized
    // auto _comm = thrust::device_vector<uint32_t>(V_MAX_IDX + 1);
    // thrust::sequence(_comm.begin(), _comm.end());
    // comm = thrust::raw_pointer_cast(&_comm[0]);


    HANDLE_ERROR(hipHostAlloc((void**)&tmp, sizeof(uint32_t) * (V_MAX_IDX + 1), hipHostMallocDefault));
    for (int i = 0; i <= V_MAX_IDX; i++) {
        tmp[i] = i; // each node is in it's own community at the beginning
    }
    HANDLE_ERROR(hipHostGetDevicePointer(&comm, tmp, 0));

    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte); // TODO customize
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(reassign_nodes), hipFuncCachePreferShared); // TODO

    hipDeviceSynchronize();

    float mod = reassign_communities(V_MAX_IDX, V, E, W, k, ac, comm, newComm, m, MIN_GAIN);

    printf("end modularity: %f\n", mod);

    // hipFree(V);
    return 0;
}
